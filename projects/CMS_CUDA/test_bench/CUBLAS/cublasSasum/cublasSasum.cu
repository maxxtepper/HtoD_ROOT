
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

int main() {
	int arraySize = 1000000;
	float *a = (float*)malloc(sizeof(float)*arraySize);
	float *d_a;
	hipMalloc((void**) &d_a, sizeof(float)*arraySize);
	for (int i=0; i<arraySize; i++) a[i] = 0.8f;
	hipMemcpy(d_a, a, sizeof(float)*arraySize, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float *cb_result = (float*)malloc(sizeof(float));

	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipblasSasum(handle, arraySize, d_a, 1, cb_result);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;

	printf("\nCUBLAS: %.3f\n\n", *cb_result);
	printf("time = %f us\n",micro);

	hipblasDestroy(handle);

	return 0;
}
