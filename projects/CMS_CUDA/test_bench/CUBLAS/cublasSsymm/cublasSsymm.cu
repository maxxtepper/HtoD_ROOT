// nvcc 037 ssymm .c -lcublas
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 333 // a - mxm matri
#define n 333 // b,c - mxn matrices
int main (void) {
	hipError_t cudaStat ; // cudaMalloc status
	hipblasStatus_t stat ; // CUBLAS functions status
	hipblasHandle_t handle ; // CUBLAS context
	int i,j; // i-row ind. , j- column ind.
	float * a; // mxm matrix a on the host
	float * b; // mxn matrix b on the host
	float * c; // mxn matrix c on the host
	a = (float*)malloc(m*m*sizeof(float)); // host memory for a
	b = (float*)malloc(m*n*sizeof(float)); // host memory for b
	c = (float*)malloc(m*n*sizeof(float)); // host memory for c
	// define the lower triangle of an mxm symmetric matrix a in
	// lower mode column by column
	int ind = 11; // a:
	for(j=0; j<m; j++){ // 11
		for(i=0; i<m; i++){ // 12 ,17
			if(i >= j){ // 13 ,18 ,22
				a[IDX2C(i,j,m)]=(float)ind++; // 14 ,19 ,23 ,26
			} // 15 ,20 ,24 ,27 ,29
		} // 16 ,21 ,25 ,28 ,30 ,31
	}
	// print the lower triangle of a row by row
//	printf (" lower triangle of a:\n");
	for (i=0;i<m;i ++){
		for (j=0;j<m;j ++){
//			if(i >=j)
//				printf (" %5.0f",a[ IDX2C (i,j,m )]);
		}
//		printf ("\n");
	}
	// define mxn matrices b,c column by column
	ind =11; // b,c:
	for(j=0;j<n;j ++){ // 11 ,17 ,23 ,29
		for(i=0;i<m;i ++){ // 12 ,18 ,24 ,30
			b[ IDX2C (i,j,m )]=( float )ind; // 13 ,19 ,25 ,31
			c[ IDX2C (i,j,m )]=( float )ind; // 14 ,20 ,26 ,32
			ind ++; // 15 ,21 ,27 ,33
		} // 16 ,22 ,28 ,34
	}
	// print b(=c) row by row
//	printf ("b(=c):\n");
	for (i=0;i<m;i ++){
		for (j=0;j<n;j ++){
//			printf (" %5.0f",b[ IDX2C (i,j,m )]);
		}
//		printf ("\n");
	}
	// on the device
	float * d_a; // d_a - a on the device
	float * d_b; // d_b - b on the device
	float * d_c; // d_c - c on the device
	cudaStat = hipMalloc (( void **)& d_a ,m*m* sizeof (*a)); // device
	// memory alloc for a
	cudaStat = hipMalloc (( void **)& d_b ,m*n* sizeof (*b)); // device
	// memory alloc for b
	cudaStat = hipMalloc (( void **)& d_c ,m*n* sizeof (*c)); // device
	// memory alloc for c
	stat = hipblasCreate (& handle ); // initialize CUBLAS context
	// copy matrices from the host to the device
	stat = hipblasSetMatrix (m,m, sizeof (*a) ,a,m,d_a ,m); //a -> d_a
	stat = hipblasSetMatrix (m,n, sizeof (*b) ,b,m,d_b ,m); //b -> d_b
	stat = hipblasSetMatrix (m,n, sizeof (*c) ,c,m,d_c ,m); //c -> d_c
	float al = 1.0f; // al =1
	float bet = 1.0f; // bet =1
	// symmetric matrix - matrix multiplication :
	// d_c = al*d_a *d_b + bet *d_c ; d_a - mxm symmetric matrix ;
	// d_b ,d_c - mxn general matrices ; al ,bet - scalars

	float milli = 0;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	stat=hipblasSsymm(handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_LOWER,
			m,n,&al,d_a,m,d_b,m,&bet,d_c,m);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;

	stat = hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); // d_c -> c
//	printf ("c after Ssymm :\n"); // print c after Ssymm
	for(i=0;i<m;i ++){
		for(j=0;j<n;j ++){
//			printf (" %7.0f",c[ IDX2C (i,j,m )]);
		}
//		printf ("\n");
	}
	printf("\ntime = %f us\n",micro);
	hipFree (d_a ); // free device memory
	hipFree (d_b ); // free device memory
	hipFree (d_c ); // free device memory
	hipblasDestroy ( handle ); // destroy CUBLAS context
	free (a); // free host memory
	free (b); // free host memory
	free (c); // free host memory
	return EXIT_SUCCESS ;
}
