#include <stdio.h>
#include "hipblas.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

int main() {
	int arraySize = 333;

	float *x, *y;
	float *d_x, *d_y;

	x = (float*)malloc(sizeof(float)*arraySize);
	y = (float*)malloc(sizeof(float)*arraySize);

	hipMalloc((void**) &d_x, sizeof(float)*arraySize);
	hipMalloc((void**) &d_y, sizeof(float)*arraySize);

	for (int i=0; i<arraySize; i++) x[i] = 0.8f;
	for (int i=0; i<arraySize; i++) y[i] = 0.6f;
	
	hipMemcpy(d_x, x, sizeof(float)*arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float)*arraySize, hipMemcpyHostToDevice);

	float *y_old;
	y_old = (float*)malloc(sizeof(float)*arraySize);
	for (int i=0; i<arraySize; i++) y_old[i] = y[i];

	float alpha = 0.4f;

	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	float milli0 = 0;
	hipEvent_t start0, stop0;
	hipEventCreate(&start0);
	hipEventCreate(&stop0);

	hipEventRecord(start0, 0);
	status = hipblasSetVector(arraySize,sizeof(float),x,1,d_x,1);
	status = hipblasSetVector(arraySize,sizeof(float),y,1,d_y,1);
	//---------------------------------------------------------------------
	status = hipblasSaxpy(handle, arraySize, &alpha, d_x, 1, d_y, 1);
	//---------------------------------------------------------------------
	status = hipblasGetVector(arraySize,sizeof(float),d_y,1,y,1);
	hipEventRecord(stop0, 0);
	hipEventSynchronize(stop0);
	hipEventElapsedTime(&milli0, start0, stop0);

	hipEventDestroy(start0);
	hipEventDestroy(stop0);
	float micro0 = milli0*1000;
	hipblasDestroy(handle);
	
	printf("%f,%f\n", y_old[0], y[0]);
	
	printf("hipblasSaxpy: Kernel = %f us\n", micro0);

	if (status) {};

	hipFree(d_x);
	hipFree(d_y);

	free(x);
	free(y);
	free(y_old);

	return 0;
}
