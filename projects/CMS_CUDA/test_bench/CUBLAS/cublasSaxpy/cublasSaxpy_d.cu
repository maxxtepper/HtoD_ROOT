#include <stdio.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

//__constant__ float alpha;
//__device__ __global__ float alpha;

int main() {
	int arraySize = 333;

	float *x, *y, *a;
	float *d_x, *d_y, *d_a;

	x = (float*)malloc(sizeof(float)*arraySize);
	y = (float*)malloc(sizeof(float)*arraySize);
	a = (float*)malloc(sizeof(float)*arraySize);

	hipMalloc((void**) &d_x, sizeof(float)*arraySize);
	hipMalloc((void**) &d_y, sizeof(float)*arraySize);
	hipMalloc((void**) &d_a, sizeof(float)*arraySize);

	for (int i=0; i<arraySize; i++) x[i] = 0.8f;
	for (int i=0; i<arraySize; i++) y[i] = 0.6f;
	for (int i=0; i<arraySize; i++) a[i] = 0.4f;
//	a[0] = 0.4f;

	hipMemcpy(d_x, x, sizeof(float)*arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float)*arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, sizeof(float)*arraySize, hipMemcpyHostToDevice);
//	cudaMemcpyToSymbol(alpha, a, sizeof(float), 0, cudaMemcpyHostToDevice);
//	cudaGetSymbolAddress((float**)d_alpha, alpha);

	float *y_old;
	y_old = (float*)malloc(sizeof(float)*arraySize);
	for (int i=0; i<arraySize; i++) y_old[i] = y[i];

	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	status = hipblasSetVector(arraySize,sizeof(float),x,1,d_x,1);
	status = hipblasSetVector(arraySize,sizeof(float),y,1,d_y,1);
	//-----------------------------------------------------------------
	status = hipblasSaxpy(handle, arraySize, d_a, d_x, 1, d_y, 1);
	//-----------------------------------------------------------------
	status = hipblasGetVector(arraySize,sizeof(float),d_y,1,y,1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;
	hipblasDestroy(handle);
	
	for (int i=0; i<arraySize; i++) {
		printf("%f,%f\n", y_old[i], y[i]);
	}
	
	printf("hipblasSaxpy time = %f us\n", micro);

	if (status){};

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_a);

	free(x);
	free(y);
	free(a);
	free(y_old);

	return 0;
}
