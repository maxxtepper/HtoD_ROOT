
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#define BYTE_MAX 28
#define ATTEMPTS 100

#define PINNED 0

const uint32_t BYTES[BYTE_MAX] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23, 1<<24, 1<<25, 1<<26, 1<<27};
const uint32_t BYTE_TOTAL[4] = {28, 27, 26, 25};

int main(int argc, char *argv[]) {
	if (argc != 3) {
		printf("Usage: HtoD_TimeByte_1a array iteration\n");
		printf("Example: HtoD_TimeByte_1a 1 1000\n");
		return 0;
	}
	int array = atoi(argv[1]);
	int iterations = atoi(argv[2]);
	printf("Starting HostToDevice Bytes vs. Time Data Collection.......\n\n");
	printf("Number of Arrays: %d\n............\n", array);
	for (int run=0; run<iterations; run++) {
		printf("\r%d/%d", run+1, iterations);
		fflush(stdout);
		//  Initialize parameters
		FILE *outFile;
		for (int i=(0+array-1); i<BYTE_TOTAL[array]; i++) {
			size_t size = BYTES[i]*sizeof(float);
			//  Declare Host variables
			float *a;
			//  Declare Device variables
			float *d_a;

			if (PINNED) {
				//  Allocate Host variables
				hipHostMalloc((void**)&a, size, hipHostMallocDefault);
			} else {
				//  Allocate Host variables
				a = (float*)malloc(size);
			}
			//  Allocate Device variables
			hipMalloc((void**)&d_a, size);
			//	Get the GPU Pointers
			//	cudaHostGetDevicePointer(&d_a, a, 0);
			//  Set Host variables
			for (uint32_t j=0; j<BYTES[j]; j++) {
				int n0 = rand()%1000000;
				int n1 = rand()%1000000;
				a[j] = n0*n1;
			}
			//  Initialize Device Event Handeling
			hipFree(0);
			float milli = 0;
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			/////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////
			//  Set Device variables
			hipEventRecord(start, 0);
			hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milli, start, stop);
			hipEventDestroy(start);
			hipEventDestroy(stop);
			//	Write Data to File
			int iBuf = i;
			char fn[50] = "rawData_bytes_1a_";
			char s_byte[50];
			snprintf(s_byte, 50, "%i", iBuf);
			strcat(fn, s_byte);
			strcat(fn, ".txt");
			outFile = fopen(fn, "a");
			char buffer_0[50];
			snprintf(buffer_0, 50, "%f", milli);
			strcat(buffer_0, ",");
			fputs(buffer_0, outFile);
			char buffer_1[50];
			snprintf(buffer_1, 50, "%i", iBuf);
			strcat(buffer_1, "\n");
			fputs(buffer_1, outFile);
			fclose(outFile);
			
			if (PINNED) {
				//  Deallocate Host variables
				hipHostFree(a);
			} else {
				//  Deallocate Host variables
				free(a);
			}
			//  Deallocate Device variables
			hipFree(d_a);
		}
	}
	printf("\n............\n\n");
	printf("Complete!!\n");
	return 0;
}
