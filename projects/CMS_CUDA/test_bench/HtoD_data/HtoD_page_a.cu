
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#define BYTE_MAX 28
#define ATTEMPTS 100

#define PINNED 0

const uint32_t BYTES[BYTE_MAX] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23, 1<<24, 1<<25, 1<<26, 1<<27};
const uint32_t BYTE_TOTAL[4] = {28, 27, 26, 25};

int main(int argc, char *argv[]) {
	//  Initialize parameters
	FILE *outFile;
	for (int i=0; i<27; i++) {
		size_t size = BYTES[i]*sizeof(float);
		//  Declare Host variables
		float *a;
		//  Declare Device variables
		float *d_a;
		//  Allocate Host variables
		a = (float*)malloc(size);
		//  Allocate Device variables
		hipMalloc(&d_a, size);
		//	Get the GPU Pointers
		//	cudaHostGetDevicePointer(&d_a, a, 0);
		//  Set Host variables
		for (uint32_t j=0; j<BYTES[i]; j++) {
			int n0 = rand()%1000000;
			int n1 = rand()%1000000;
			a[j] = n0*n1;
		}
		//  Initialize Device Event Handeling
		hipFree(0);
		float milli = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		/////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////
		//  Set Device variables
		hipEventRecord(start, 0);
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		//	Write Data to File
		int iBuf = i;
		char fn[50] = "rawData_bytes_1a_";
		char s_byte[50];
		snprintf(s_byte, 50, "%i", iBuf);
		strcat(fn, s_byte);
		strcat(fn, ".txt");
		outFile = fopen(fn, "a");
		char buffer_0[50];
		snprintf(buffer_0, 50, "%f", milli);
		strcat(buffer_0, ",");
		fputs(buffer_0, outFile);
		char buffer_1[50];
		snprintf(buffer_1, 50, "%i", iBuf);
		strcat(buffer_1, "\n");
		fputs(buffer_1, outFile);
		fclose(outFile);
		//  Deallocate Host variables
		free(a);
		//  Deallocate Device variables
		hipFree(d_a);
	}
	return 0;
}
