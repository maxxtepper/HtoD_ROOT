
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#define BYTE_TOTAL 24
#define ATTEMPTS 100

int main (void) {
//	printf("Starting HostToDevice Bytes vs. Time Data Collection.......\n");
	//  Initialize parameters
	const uint32_t BYTES[BYTE_TOTAL] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23};
	FILE *outFile;
	for (int i=0; i<BYTE_TOTAL; i++) {
//		printf("Beginning Byte Total 2^%i.......\n", i);
		size_t size = BYTES[i]*sizeof(float);
		//  Declare Host variables
		float *a;
		float *b;
		float *c;
		float *d;
		float *e;
		float *f;
		float *g;
		float *h;
		float *aa;
		float *ba;
		float *ca;
		float *da;
		float *ea;
		float *fa;
		float *ga;
		float *ha;
		//  Declare Device variables
		float *d_a;
		float *d_b;
		float *d_c;
		float *d_d;
		float *d_e;
		float *d_f;
		float *d_g;
		float *d_h;
		float *d_aa;
		float *d_ba;
		float *d_ca;
		float *d_da;
		float *d_ea;
		float *d_fa;
		float *d_ga;
		float *d_ha;
		//  Allocate Host variables
		hipHostAlloc((void**)&a, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&b, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&c, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&d, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&e, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&f, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&g, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&h, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&aa, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&ba, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&ca, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&da, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&ea, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&fa, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&ga, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&ha, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		//  Allocate Device variables
		hipMalloc(&d_a, size);
		hipMalloc(&d_b, size);
		hipMalloc(&d_c, size);
		hipMalloc(&d_d, size);
		hipMalloc(&d_e, size);
		hipMalloc(&d_f, size);
		hipMalloc(&d_g, size);
		hipMalloc(&d_h, size);
		hipMalloc(&d_aa, size);
		hipMalloc(&d_ba, size);
		hipMalloc(&d_ca, size);
		hipMalloc(&d_da, size);
		hipMalloc(&d_ea, size);
		hipMalloc(&d_fa, size);
		hipMalloc(&d_ga, size);
		hipMalloc(&d_ha, size);
		//	Get the GPU Pointers
		hipHostGetDevicePointer(&d_a, a, 0);
		hipHostGetDevicePointer(&d_b, b, 0);
		hipHostGetDevicePointer(&d_c, c, 0);
		hipHostGetDevicePointer(&d_d, d, 0);
		hipHostGetDevicePointer(&d_e, a, 0);
		hipHostGetDevicePointer(&d_f, b, 0);
		hipHostGetDevicePointer(&d_g, c, 0);
		hipHostGetDevicePointer(&d_h, d, 0);
		hipHostGetDevicePointer(&d_aa, aa, 0);
		hipHostGetDevicePointer(&d_ba, ba, 0);
		hipHostGetDevicePointer(&d_ca, ca, 0);
		hipHostGetDevicePointer(&d_da, da, 0);
		hipHostGetDevicePointer(&d_ea, ea, 0);
		hipHostGetDevicePointer(&d_fa, fa, 0);
		hipHostGetDevicePointer(&d_ga, ga, 0);
		hipHostGetDevicePointer(&d_ha, ha, 0);
		//  Set Host variables
		for (uint32_t j=0; j<BYTES[j]; j++) {
			int n0 = rand()%1000000;
			int n1 = rand()%1000000;
			int n2 = rand()%1000000;
			int n3 = rand()%1000000;
			int n4 = rand()%1000000;
			int n5 = rand()%1000000;
			int n6 = rand()%1000000;
			int n7 = rand()%1000000;
			int n8 = rand()%1000000;
			int n9 = rand()%1000000;
			int n10 = rand()%1000000;
			int n11 = rand()%1000000;
			int n12 = rand()%1000000;
			int n13 = rand()%1000000;
			int n14 = rand()%1000000;
			int n15 = rand()%1000000;
			int n0a = rand()%1000000;
			int n1a = rand()%1000000;
			int n2a = rand()%1000000;
			int n3a = rand()%1000000;
			int n4a = rand()%1000000;
			int n5a = rand()%1000000;
			int n6a = rand()%1000000;
			int n7a = rand()%1000000;
			int n8a = rand()%1000000;
			int n9a = rand()%1000000;
			int n10a = rand()%1000000;
			int n11a = rand()%1000000;
			int n12a = rand()%1000000;
			int n13a = rand()%1000000;
			int n14a = rand()%1000000;
			int n15a = rand()%1000000;
			a[j] = n0*n1;
			b[j] = n2*n3;
			c[j] = n4*n5;
			d[j] = n5*n6;
			e[j] = n7*n8;
			f[j] = n9*n10;
			g[j] = n11*n12;
			h[j] = n13*n14;
			aa[j] = n0a*n1a;
			ba[j] = n2a*n3a;
			ca[j] = n4a*n5a;
			da[j] = n5a*n6a;
			ea[j] = n7a*n8a;
			fa[j] = n9a*n10a;
			ga[j] = n11a*n12a;
			ha[j] = n13a*n14a;
		}
		//  Initialize Device Event Handeling
		float milli = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		/////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////
		//  Set Device variables
		hipEventRecord(start);
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
		hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
		hipMemcpy(d_e, e, size, hipMemcpyHostToDevice);
		hipMemcpy(d_f, f, size, hipMemcpyHostToDevice);
		hipMemcpy(d_g, g, size, hipMemcpyHostToDevice);
		hipMemcpy(d_h, h, size, hipMemcpyHostToDevice);
		hipMemcpy(d_aa, aa, size, hipMemcpyHostToDevice);
		hipMemcpy(d_ba, ba, size, hipMemcpyHostToDevice);
		hipMemcpy(d_ca, ca, size, hipMemcpyHostToDevice);
		hipMemcpy(d_da, da, size, hipMemcpyHostToDevice);
		hipMemcpy(d_ea, ea, size, hipMemcpyHostToDevice);
		hipMemcpy(d_fa, fa, size, hipMemcpyHostToDevice);
		hipMemcpy(d_ga, ga, size, hipMemcpyHostToDevice);
		hipMemcpy(d_ha, ha, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		//	Write Data to File
		int iBuf = i+4;
		char fn[50] = "rawData_bytes_16a_";
		char s_byte[50];
		snprintf(s_byte, 50, "%i", iBuf);
		strcat(fn, s_byte);
		strcat(fn, ".txt");
		outFile = fopen(fn, "a");
		char buffer_0[50];
		snprintf(buffer_0, 50, "%f", milli);
		strcat(buffer_0, ",");
		fputs(buffer_0, outFile);
		char buffer_1[50];
		snprintf(buffer_1, 50, "%i", iBuf);
		strcat(buffer_1, "\n");
		fputs(buffer_1, outFile);
		fclose(outFile);
		//  Deallocate Host variables
		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);
		hipHostFree(d);
		hipHostFree(e);
		hipHostFree(f);
		hipHostFree(g);
		hipHostFree(h);
		hipHostFree(aa);
		hipHostFree(ba);
		hipHostFree(ca);
		hipHostFree(da);
		hipHostFree(ea);
		hipHostFree(fa);
		hipHostFree(ga);
		hipHostFree(ha);
		//  Deallocate Device variables
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
		hipFree(d_e);
		hipFree(d_f);
		hipFree(d_g);
		hipFree(d_h);
		hipFree(d_aa);
		hipFree(d_ba);
		hipFree(d_ca);
		hipFree(d_da);
		hipFree(d_ea);
		hipFree(d_fa);
		hipFree(d_ga);
		hipFree(d_ha);
	}
	return 0;
}
