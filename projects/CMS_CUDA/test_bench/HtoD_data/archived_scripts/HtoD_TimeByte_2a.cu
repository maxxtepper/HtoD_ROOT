
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#define BYTE_TOTAL 27
#define ATTEMPTS 100

int main (void) {
//	printf("Starting HostToDevice Bytes vs. Time Data Collection.......\n");
	//  Initialize parameters
	const uint32_t BYTES[BYTE_TOTAL] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23, 1<<24, 1<<25, 1<<26};
	FILE *outFile;
	for (int i=0; i<BYTE_TOTAL; i++) {
//		printf("Beginning Byte Total 2^%i.......\n", i);
		size_t size = BYTES[i]*sizeof(float);
		//  Declare Host variables
		float *a;
		float *b;
		//  Declare Device variables
		float *d_a;
		float *d_b;
		//  Allocate Host variables
		hipHostAlloc((void**)&a, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&b, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		//  Allocate Device variables
		hipMalloc(&d_a, size);
		hipMalloc(&d_b, size);
		//	Get the GPU Pointers
		hipHostGetDevicePointer(&d_a, a, 0);
		hipHostGetDevicePointer(&d_b, b, 0);
		//  Set Host variables
		for (uint32_t j=0; j<BYTES[j]; j++) {
			int n0 = rand()%1000000;
			int n1 = rand()%1000000;
			int n2 = rand()%1000000;
			int n3 = rand()%1000000;
			a[j] = n0*n1;
			b[j] = n2*n3;
		}
		//  Initialize Device Event Handeling
		float milli = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		/////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////
		//  Set Device variables
		hipEventRecord(start);
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		//	Write Data to File
		int iBuf = i+1;
		char fn[50] = "rawData_bytes_2a_";
		char s_byte[50];
		snprintf(s_byte, 50, "%i", iBuf);
		strcat(fn, s_byte);
		strcat(fn, ".txt");
		outFile = fopen(fn, "a");
		char buffer_0[50];
		snprintf(buffer_0, 50, "%f", milli);
		strcat(buffer_0, ",");
		fputs(buffer_0, outFile);
		char buffer_1[50];
		snprintf(buffer_1, 50, "%i", iBuf);
		strcat(buffer_1, "\n");
		fputs(buffer_1, outFile);
		fclose(outFile);
		//  Deallocate Host variables
		hipHostFree(a);
		hipHostFree(b);
		//  Deallocate Device variables
		hipFree(d_a);
		hipFree(d_b);
	}
	return 0;
}
