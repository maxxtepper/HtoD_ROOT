
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#define BYTE_TOTAL 26
#define ATTEMPTS 100

int main (void) {
//	printf("Starting HostToDevice Bytes vs. Time Data Collection.......\n");
	//  Initialize parameters
	const uint32_t BYTES[BYTE_TOTAL] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23, 1<<24, 1<<25};
	FILE *outFile;
	for (int i=0; i<BYTE_TOTAL; i++) {
//		printf("Beginning Byte Total 2^%i.......\n", i);
		size_t size = BYTES[i]*sizeof(float);
		//  Declare Host variables
		float *a;
		float *b;
		float *c;
		float *d;
		//  Declare Device variables
		float *d_a;
		float *d_b;
		float *d_c;
		float *d_d;
		//  Allocate Host variables
		hipHostAlloc((void**)&a, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&b, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&c, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&d, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		//  Allocate Device variables
		hipMalloc(&d_a, size);
		hipMalloc(&d_b, size);
		hipMalloc(&d_c, size);
		hipMalloc(&d_d, size);
		//	Get the GPU Pointers
		hipHostGetDevicePointer(&d_a, a, 0);
		hipHostGetDevicePointer(&d_b, b, 0);
		hipHostGetDevicePointer(&d_c, c, 0);
		hipHostGetDevicePointer(&d_d, d, 0);
		//  Set Host variables
		for (uint32_t j=0; j<BYTES[j]; j++) {
			int n0 = rand()%1000000;
			int n1 = rand()%1000000;
			int n2 = rand()%1000000;
			int n3 = rand()%1000000;
			int n4 = rand()%1000000;
			int n5 = rand()%1000000;
			int n6 = rand()%1000000;
			int n7 = rand()%1000000;
			a[j] = n0*n1;
			b[j] = n2*n3;
			c[j] = n4*n5;
			d[j] = n5*n6;
		}
		//  Initialize Device Event Handeling
		float milli = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		/////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////
		//  Set Device variables
		hipEventRecord(start);
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
		hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		//	Write Data to File
		int iBuf = i+2;
		char fn[50] = "rawData_bytes_4a_";
		char s_byte[50];
		snprintf(s_byte, 50, "%i", iBuf);
		strcat(fn, s_byte);
		strcat(fn, ".txt");
		outFile = fopen(fn, "a");
		char buffer_0[50];
		snprintf(buffer_0, 50, "%f", milli);
		strcat(buffer_0, ",");
		fputs(buffer_0, outFile);
		char buffer_1[50];
		snprintf(buffer_1, 50, "%i", iBuf);
		strcat(buffer_1, "\n");
		fputs(buffer_1, outFile);
		fclose(outFile);
		//  Deallocate Host variables
		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);
		hipHostFree(d);
		//  Deallocate Device variables
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
	}
	return 0;
}
