
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#define BYTE_TOTAL 25
#define ATTEMPTS 100

int main (void) {
//	printf("Starting HostToDevice Bytes vs. Time Data Collection.......\n");
	//  Initialize parameters
	const uint32_t BYTES[BYTE_TOTAL] = {1<<0, 1<<1, 1<<2, 1<<3, 1<<4, 1<<5, 1<<6, 1<<7, 1<<8, 1<<9, 1<<10, 1<<11, 1<<12, 1<<13, 1<<14, 1<<15, 1<<16, 1<<17, 1<<18, 1<<19, 1<<20, 1<<21, 1<<22, 1<<23, 1<<24};
	FILE *outFile;
	for (int i=0; i<BYTE_TOTAL; i++) {
//		printf("Beginning Byte Total 2^%i.......\n", i);
		size_t size = BYTES[i]*sizeof(float);
		//  Declare Host variables
		float *a;
		float *b;
		float *c;
		float *d;
		float *e;
		float *f;
		float *g;
		float *h;
		//  Declare Device variables
		float *d_a;
		float *d_b;
		float *d_c;
		float *d_d;
		float *d_e;
		float *d_f;
		float *d_g;
		float *d_h;
		//  Allocate Host variables
		hipHostAlloc((void**)&a, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&b, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&c, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&d, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&e, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&f, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&g, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		hipHostAlloc((void**)&h, size, hipHostMallocWriteCombined | hipHostMallocMapped);
		//  Allocate Device variables
		hipMalloc(&d_a, size);
		hipMalloc(&d_b, size);
		hipMalloc(&d_c, size);
		hipMalloc(&d_d, size);
		hipMalloc(&d_e, size);
		hipMalloc(&d_f, size);
		hipMalloc(&d_g, size);
		hipMalloc(&d_h, size);
		//	Get the GPU Pointers
		hipHostGetDevicePointer(&d_a, a, 0);
		hipHostGetDevicePointer(&d_b, b, 0);
		hipHostGetDevicePointer(&d_c, c, 0);
		hipHostGetDevicePointer(&d_d, d, 0);
		hipHostGetDevicePointer(&d_e, e, 0);
		hipHostGetDevicePointer(&d_f, f, 0);
		hipHostGetDevicePointer(&d_g, g, 0);
		hipHostGetDevicePointer(&d_h, h, 0);
		//  Set Host variables
		for (uint32_t j=0; j<BYTES[j]; j++) {
			int n0 = rand()%1000000;
			int n1 = rand()%1000000;
			int n2 = rand()%1000000;
			int n3 = rand()%1000000;
			int n4 = rand()%1000000;
			int n5 = rand()%1000000;
			int n6 = rand()%1000000;
			int n7 = rand()%1000000;
			int n8 = rand()%1000000;
			int n9 = rand()%1000000;
			int n10 = rand()%1000000;
			int n11 = rand()%1000000;
			int n12 = rand()%1000000;
			int n13 = rand()%1000000;
			int n14 = rand()%1000000;
			int n15 = rand()%1000000;
			a[j] = n0*n1;
			b[j] = n2*n3;
			c[j] = n4*n5;
			d[j] = n5*n6;
			e[j] = n7*n8;
			f[j] = n9*n10;
			g[j] = n11*n12;
			h[j] = n13*n14;
		}
		//  Initialize Device Event Handeling
		float milli = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		/////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////
		//  Set Device variables
		hipEventRecord(start);
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
		hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
		hipMemcpy(d_e, e, size, hipMemcpyHostToDevice);
		hipMemcpy(d_f, f, size, hipMemcpyHostToDevice);
		hipMemcpy(d_g, g, size, hipMemcpyHostToDevice);
		hipMemcpy(d_h, h, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		//	Write Data to File
		int iBuf = i+3;
		char fn[50] = "rawData_bytes_8a_";
		char s_byte[50];
		snprintf(s_byte, 50, "%i", iBuf);
		strcat(fn, s_byte);
		strcat(fn, ".txt");
		outFile = fopen(fn, "a");
		char buffer_0[50];
		snprintf(buffer_0, 50, "%f", milli);
		strcat(buffer_0, ",");
		fputs(buffer_0, outFile);
		char buffer_1[50];
		snprintf(buffer_1, 50, "%i", iBuf);
		strcat(buffer_1, "\n");
		fputs(buffer_1, outFile);
		fclose(outFile);
		//  Deallocate Host variables
		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);
		hipHostFree(d);
		hipHostFree(e);
		hipHostFree(f);
		hipHostFree(g);
		hipHostFree(h);
		//  Deallocate Device variables
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
		hipFree(d_e);
		hipFree(d_f);
		hipFree(d_g);
		hipFree(d_h);
	}
	return 0;
}
