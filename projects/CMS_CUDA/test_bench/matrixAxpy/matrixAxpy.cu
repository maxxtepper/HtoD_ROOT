#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//__global__ void Mat_add(double A[], double B[], double C[], double alpha[], int m, int n) {
__global__ void Mat_add(double *A, double *B, double *C, double *alpha, int m, int n) {
	int my_ij = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < m && threadIdx.x < n) 
		C[my_ij] = A[my_ij] - alpha[0]*B[my_ij];
} 

/* Host code */
int main(int argc, char* argv[]) {
	int m, n;
	double *h_A, *h_B, *h_C, *h_alpha;
	double *d_A, *d_B, *d_C, *d_alpha;
	size_t size;

	/* Get size of matrices */
	if (argc != 3) {
		fprintf(stderr, "usage: %s <row count> <col count>\n", argv[0]);
		exit(0);
	}
	m = strtol(argv[1], NULL, 10);
	n = strtol(argv[2], NULL, 10);
	printf("m = %d, n = %d\n", m, n);
	size = m*n*sizeof(double);

	h_A     = (double*)malloc(size);
	h_B     = (double*)malloc(size);
	h_C     = (double*)malloc(size);
	h_alpha = (double*)malloc(size);

	for (int i = 0; i < m; i++)
		for (int j = 0; j < n; j++) {
			h_A[i*n+j] = 2.0f;
			h_B[i*n+j] = 4.0f;
			h_C[i*n+j] = 0.0f;
			h_alpha[i*n+j] = 6.0f;
		}

	/* Allocate matrices in device memory */
	hipMalloc(&d_A    , size);
	hipMalloc(&d_B    , size);
	hipMalloc(&d_C    , size);
	hipMalloc(&d_alpha, size);

	/* Copy matrices from host memory to device memory */
	hipMemcpy(d_A    , h_A    , size, hipMemcpyHostToDevice);
	hipMemcpy(d_B    , h_B    , size, hipMemcpyHostToDevice);
	hipMemcpy(d_alpha, h_alpha, size, hipMemcpyHostToDevice);

	/* Invoke kernel using m thread blocks, each of    */
	/* which contains n threads                        */
	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	Mat_add<<<m, n>>>(d_A, d_B, d_C, d_alpha, m, n);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);

	/* Wait for the kernel to complete */
	hipDeviceSynchronize();

	/* Copy result from device memory to host memory */
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

//	for (int i = 0; i < m; i++)
//		for (int j = 0; j < n; j++) 
//			printf("%f   %f   %f\n", h_A[i*n+j], h_B[i*n+j], h_C[i*n+j]);
	printf("%f ms\n", milli);
	/* Free device memory */
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_alpha);

	/* Free host memory */
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_alpha);

	return 0;
}  /* main */
