
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
	unsigned int N = 450000000;
	unsigned int bytes = N*sizeof(double);
	//	Host Initialization
	double *h_a;
	h_a = (double*)malloc(bytes);
	for (unsigned int i=0; i<N; i++)
		h_a[i] = 2.0f;

	//	Device Initialization
	double *d_a;
	hipMalloc(&d_a, bytes);
	
	//	Event Initialization
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milli = 0;

	hipEventRecord(start,0);
	hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);

	printf("%f ms\n",milli);

	//	Cleanup
	free(h_a);
	hipFree(d_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
