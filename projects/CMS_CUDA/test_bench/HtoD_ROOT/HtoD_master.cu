#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#include "HtoD_master.h"

extern float HtoD(const uint32_t dim, bool pinned) {
	//  Initialize parameters
	size_t size = dim*sizeof(float);
	//  Declare Host variables
	float *a;
	//  Declare Device variables
	float *d_a;
	//  Allocate Host variables
	if (pinned)
		hipHostMalloc((void**)&a, size);
	else
		a = (float*)malloc(size);
	//  Allocate Device variables
	hipMalloc((void**)&d_a, size);
	//  Set Host variables
	for (uint32_t j=0; j<dim; j++) {
		int n0 = rand()%1000000;
		int n1 = rand()%1000000;
		a[j] = n0*n1;
	}
	//  Initialize Device Event Handeling
	hipFree(0);
	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	/////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////
	//  Set Device variables
	hipEventRecord(start, 0);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//  Deallocate Host variables
	if (pinned)
		hipHostFree(a);
	else
		free(a);
	//  Deallocate Device variables
	hipFree(d_a);
	return milli;
}
