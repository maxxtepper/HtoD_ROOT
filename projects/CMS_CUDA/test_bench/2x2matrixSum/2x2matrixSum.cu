
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

typedef struct {
	long width;
	long height;
	float *element;
} Matrix;

__global__ void sum(long N, const Matrix A, const Matrix B, Matrix C) {
	int i = threadIdx.x;

	if (i<N) {
		C.element[i] = A.element[i]+B.element[i];
	}
}

int main(void) {
	long DIM_0 = 1<<10;
	long DIM_1 = 1<<10;
	//	CUDA Performance Variables
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//	Declare initial parameters
	long N = DIM_0*DIM_1;
	size_t size = N*sizeof(float);
	//	Create Host variables
	Matrix mA = {DIM_0, DIM_1};
	Matrix mB = {DIM_0, DIM_1};
	Matrix mC = {DIM_0, DIM_1};
	//	Create Device variables
	Matrix d_mA = {DIM_0, DIM_1};
	Matrix d_mB = {DIM_0, DIM_1};
	Matrix d_mC = {DIM_0, DIM_1};
	//	Allocate memory for Host
	mA.element = (float*)malloc(size);
	mB.element = (float*)malloc(size);
	mC.element = (float*)malloc(size);
	//	Set Host values
	for (long i=0; i<(DIM_0*DIM_1); i++) {
		mA.element[i] = 1;
		mB.element[i] = 1;
	}
	//	Allocate memory for Device
	hipMalloc(&d_mA.element, size);
	hipMalloc(&d_mB.element, size);
	hipMalloc(&d_mC.element, size);
	//	Set Device values
	hipMemcpy(d_mA.element, mA.element, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mB.element, mB.element, size, hipMemcpyHostToDevice);
	//	Send kernel to Device
	dim3 threadsPerBlock(256);
	dim3 blocksPerGrid((N-1+256)/255);
	hipEventRecord(start);
	sum<<<blocksPerGrid, threadsPerBlock>>>(N, d_mA, d_mB, d_mC);
	//sum<<<1, 4>>>(N, d_mA, d_mB, d_mC);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	float microseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	microseconds = milliseconds*1000;
	//	Retrieve Device variables
	hipMemcpy(mC.element, d_mC.element, size, hipMemcpyDeviceToHost);
	//	Print results
	printf("%f    %f\n", mC.element[0], mC.element[1]);
	printf("%f    %f\n", mC.element[2], mC.element[3]);
	printf("Time taken (microseconds): %f\n", microseconds);
	free(mA.element);
	free(mB.element);
	free(mC.element);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

