#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__
void saxpy(int n, double a[], double x[], double y[]) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a[i]*x[i] + y[i];
}

int main(void) {
	//int N = 1<<20;
	//int N = 1<<8;
	int N = 333;
	//  Host Variables (CPU)
	double *x, *y, *a;
	//  Device Variables (GPU)
	double *d_x, *d_y, *d_a;
	//  Allocate memory in Host
	x = (double*)malloc(N*sizeof(double));
	y = (double*)malloc(N*sizeof(double));
	a = (double*)malloc(N*sizeof(double));
	//  Allocate memory in Device
	hipMalloc(&d_x, N*sizeof(double));
	hipMalloc(&d_y, N*sizeof(double));
	hipMalloc(&d_a, N*sizeof(double));
	//  Set Host variable values
	for (int i=0; i<N; i++) {
		x[i] = rand() % 1<<20;
		y[i] = rand() % 1<<20;
		a[i] = 2.0f;
	}
	//  Send Host variables to Device variables
	hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, N*sizeof(double), hipMemcpyHostToDevice);
	//  Send kernel to Device 

	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

//	saxpy<<<(N+255)/256, 256>>>(N, d_a, d_x, d_y);
	saxpy<<<1,333>>>(N, d_a, d_x, d_y);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;

	//  Retrieve Device variables
	hipMemcpy(y, d_y, N*sizeof(double), hipMemcpyDeviceToHost);
	//  Calculate error
	double maxError = 0.0f;
	for (int i=0; i<N; i++) {
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	printf("time = %f us\n",micro);


	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_a);
	free(x);
	free(y);
	free(a);
}
