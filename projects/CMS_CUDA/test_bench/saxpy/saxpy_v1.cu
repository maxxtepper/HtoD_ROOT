#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

int main(void) {
	//int N = 1<<20;
	//int N = 1<<8;
	int N = 333;
	//  Host Variables (CPU)
	float *x, *y;
	//  Device Variables (GPU)
	float *d_x, *d_y;
	//  Allocate memory in Host
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	//  Allocate memory in Device
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	//  Set Host variable values
	for (int i=0; i<N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	//  Send Host variables to Device variables
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	//  Send kernel to Device 

	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;

	//  Retrieve Device variables
	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
	//  Calculate error
	float maxError = 0.0f;
	for (int i=0; i<N; i++) {
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	printf("time = %f us\n",micro);
}
