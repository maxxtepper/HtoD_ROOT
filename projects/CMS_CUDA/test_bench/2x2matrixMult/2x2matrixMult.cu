#include "hip/hip_runtime.h"
#include <stdio.h>

#define DIM_0 2
#define DIM_1 2
#define DIM_2 2

__global__ void product(int N, float *A[N][N], float *B[N][N], float *C[N][N][N], float *D[N][N]) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int k = threadIdx.z;

	if (i<N && j<N) {
		__shared__ C[i][j][k] = A[i][k]*B[k][j];
	}
	////////////////
	__syncthreads();
	////////////////
	for (int m=0; m<N; m++) {
		D[i][j]+=C[i][j][m];
	}
}

int main(void) {
	//	Declare initial parameters
	int N = DIM_0*DIM_1;
	size_t size = N*sizeof(float);
	//	Create Host variables
	float *mA, *mB, *mC, *mD;
	//	Create Device variables
	float *d_mA, *d_mB, *d_mC, *d_mD;
	//	Allocate memory for Host
	mA = (float*)malloc(size);
	mB = (float*)malloc(size);
	mC = (float*)malloc(size*2);
	mD = (float*)malloc(size);
	//	Set Host values
	for (int i=0; i<4; i++) {
		mA[i] = (i+1);
		mB[i] = (i+5);
	}
	//	Allocate memory for Device
	hipMalloc(&d_mA, size);
	hipMalloc(&d_mB, size);
	hipMalloc(&d_mC, size*2);
	hipMalloc(&d_mD, size);
	//	Set Device values
	hipMemcpy(d_mA, mA, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mB, mB, size, hipMemcpyHostToDevice);
	//	Send kernel to Device
	dim3 threadsPerBlock(DIM_0, DIM_1, DIM_2);
	product<<<1, threadsPerBlock>>>(size*2, d_mA, d_mB, d_mC, d_mD);
	//	Retrieve Device variables
	hipMemcpy(mD, d_mD, size, hipMemcpyDeviceToHost);
	//	Print results
	printf("%f    %f\n", mD[0], mD[1]);
	printf("%f    %f\n", mD[2], mD[3]);;
	return 0;
}

