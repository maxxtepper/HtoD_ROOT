
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void inverse(int n, double *x, double *y, double *z) {
	int i = threadIdx.x;
	if (i < n) z[0] = 1/(x[0]+y[0]);
}

int main(void) {
	int N = 333;
	int bytes = N*sizeof(double);

	double *x, *y, *z;
	double *d_x, *d_y, *d_z;

	x = (double*)malloc(bytes);
	y = (double*)malloc(bytes);
	z = (double*)malloc(bytes);

	hipMalloc(&d_x, bytes);
	hipMalloc(&d_y, bytes);
	hipMalloc(&d_z, bytes);

	x[0] = 1.0f;
	y[0] = 2.0f;

	hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

	float milli = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	inverse<<<1,1>>>(N, d_x, d_y, d_z);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	float micro = milli*1000;

	hipMemcpy(z, d_z, bytes, hipMemcpyDeviceToHost);

	double answer = 1/(x[0]+y[0]);

	printf("answer = %f\n", answer);
	printf("z      = %f\n", z[0]);
	printf("time   = %f us\n",micro);

	free(x);
	free(y);
	free(z);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	return 0;
}
