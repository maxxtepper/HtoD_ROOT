#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nanoTime.h"

__global__ void productAB(int N, float *a, float *b, float *c);

__global__ void sumAB(int N, float *c, float *d);

__global__ void test(int N, float *c, float *d);

int main (void) {
		printf("2x2matrixMult_v0\n");
    //  Initialize parameters
		int N = 2<<12;
    size_t bytes = N*sizeof(float);
    //  Declare Host variables
    float *a, *b, *c, *d;
    //  Declare Device variables
    float *d_a, *d_b, *d_c, *d_d;
    //  Allocate Host variables
    a = (float*)malloc(bytes);
    b = (float*)malloc(bytes);
    c = (float*)malloc(bytes);
    d = (float*)malloc(bytes);
    //  Allocate Device variables
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    hipMalloc(&d_d, bytes);
    //  Set Host variables
		for (int i=0; i<N; i++) {
			a[i] = 1.0f;
			b[i] = 5.0f;
		}
    //  Set Device variables
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, bytes, hipMemcpyHostToDevice);
    //  Initialize Host Event Handeling
    struct timespec vartime;
    float time_elapsed_nanos;
    //  Initialize Device Event Handeling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /////////////////////////////////////////////////////////
    ////  GPU Calculation  //////////////////////////////////

		dim3 threadsPerBlock(8,8);
		dim3 numBlocks(N/threadsPerBlock.x,N/threadsPerBlock.y);

		productAB<<<numBlocks,threadsPerBlock>>>(N, d_a, d_b, d_c);
		sumAB<<<1,4>>>(N, d_c, d_d);

		float milli0 = 0;
		float micro0 = 0;
		float milli1 = 0;
		float micro1 = 0;
		float milli2 = 0;
		float micro2 = 0;
		hipStream_t stream0, stream1, stream2;
		hipStreamCreate(&stream0);
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);

		hipEventRecord(start,stream0);
		productAB<<<numBlocks,threadsPerBlock,0,stream0>>>(N, d_a, d_b, d_c);
		hipEventRecord(stop,stream0);
		hipStreamSynchronize(stream0);
		hipEventElapsedTime(&milli0, start, stop);
		micro0 = milli0*1000;

		hipEventRecord(start,stream1);
		sumAB<<<32,100,0,stream1>>>(N, d_c, d_d);
		hipEventRecord(stop,stream1);
		hipStreamSynchronize(stream1);
		hipEventElapsedTime(&milli1, start, stop);
		micro1 = milli1*1000;

		hipEventRecord(start,stream2);
		test<<<1,4,0,stream2>>>(N, d_c, d_d);
		hipEventRecord(stop,stream2);
		hipStreamSynchronize(stream2);
		hipEventElapsedTime(&milli2, start, stop);
		micro2 = (milli2*1000);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);

    vartime = timer_start();
		productAB<<<1,8>>>(N, d_a, d_b, d_c);
		sumAB<<<1,4>>>(N, d_c, d_d);
    time_elapsed_nanos = timer_end(vartime);
		float time_elapsed_micro = time_elapsed_nanos/1000;

    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Get Device variables
    hipMemcpy(d, d_d, bytes, hipMemcpyDeviceToHost);
    //  Results (GPU)
    printf("GPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Kernel Prod Elapsed Time (microseconds): %f\n", micro0);
    printf("Kernel Sum  Elapsed Time (microseconds): %f\n", micro1);
    printf("Kernel Test Elapsed Time (microseconds): %f\n", micro2);
    /////////////////////////////////////////////////////////
    ////  CPU Calculation  //////////////////////////////////
    //vartime = timer_start();
    for (int i=0; i<8; i++) c[i] = a[i]*b[i];
    for (int i=0; i<4; i++) d[i] = c[i]+c[i+4];
    //time_elapsed_nanos = timer_end(vartime);
		//float time_elapsed_micro = time_elapsed_nanos/1000;
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Results (CPU)
    //printf("CPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    //printf("Elapsed Time (microseconds): %f\n", time_elapsed_micro);
    //  Deallocate Host variables
    free(a);
    free(b);
    free(c);
    free(d);
    //  Deallocate Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    return 0;
}

__global__ void productAB(int N, float *a, float *b, float *c) {
    //int i = threadIdx.x;
		int i = (blockIdx.x*blockDim.x) + threadIdx.x;
		int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    if (i<N) c[i] = a[i]*b[j];
}

__global__ void sumAB(int N, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) d[i] = c[i]+c[i+(N/2)];
}

__global__ void test(int N, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) d[i] = (1/(c[i]+3)); 
}
