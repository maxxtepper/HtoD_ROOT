#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nanoTime.h"

__global__ void product_sumAB(int N, float *a, float *b, float *c, float *d, float *e);

int main (void) {
    //  Initialize parameters
		printf("2x2matrixMult_v1\n");
    int DIM_0 = 2;
    int DIM_1 = 2;
    int N = DIM_0*DIM_1;
    size_t size = N*sizeof(float);
    //  Declare Host variables
    float *a, *b, *c, *d, *e;
    //  Declare Device variables
    float *d_a, *d_b, *d_c, *d_d, *d_e;
    //  Allocate Host variables
    a = (float*)malloc(size*2);
    b = (float*)malloc(size*2);
    c = (float*)malloc(size*2);
    d = (float*)malloc(size);
    e = (float*)malloc(size);
    //  Allocate Device variables
    hipMalloc(&d_a, size*2);
    hipMalloc(&d_b, size*2);
    hipMalloc(&d_c, size*2);
    hipMalloc(&d_d, size);
    hipMalloc(&d_e, size);
    //  Set Host variables
    a[0] = 1;
    a[1] = 1;
    a[2] = 3;
    a[3] = 3;
    a[4] = 2;
    a[5] = 2;
    a[6] = 4;
    a[7] = 4;
    b[0] = 5;
    b[1] = 6;
    b[2] = 5;
    b[3] = 6;
    b[4] = 7;
    b[5] = 8;
    b[6] = 7;
    b[7] = 8;
    //  Set Device variables
    hipMemcpy(d_a, a, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*2, hipMemcpyHostToDevice);
    //hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
    //hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
    hipMemcpy(d_e, e, size, hipMemcpyHostToDevice);
    //  Initialize Host Event Handeling
    struct timespec vartime;
    float time_elapsed_nanos;
    //  Initialize Device Event Handeling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /////////////////////////////////////////////////////////
    ////  GPU Calculation  //////////////////////////////////
		float milli = 0;
		float micro = 0;
//		for (int i=0; i<100000; i++) {
			//  Kernel 0 (Products and Sums)
			hipEventRecord(start);
			product_sumAB<<<1,4>>>(N, d_a, d_b, d_c, d_d, d_e);
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			//  Device Event Results
			hipEventElapsedTime(&milli, start, stop);
			micro += (milli*1000);
//		}
//		micro /= 100000;
    //  Kernels Complete
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Get Device variables
    hipMemcpy(e, d_e, size, hipMemcpyDeviceToHost);
    //  Results (GPU)
    printf("GPU Calculation:\n");
    //printf("%f     %f\n", e[0], e[1]);
    //printf("%f     %f\n", e[2], e[3]);
    printf("Elapsed Time (microseconds): %f\n", micro);
    /////////////////////////////////////////////////////////
    ////  CPU Calculation  //////////////////////////////////
    vartime = timer_start();
    for (int i=0; i<8; i++) c[i] = a[i]*b[i];
    for (int i=0; i<4; i++) d[i] = c[i]+c[i+4];
    time_elapsed_nanos = timer_end(vartime);
		float time_elapsed_micro = time_elapsed_nanos/1000;
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Results (CPU)
    printf("CPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Elapsed Time (microseconds): %f\n", time_elapsed_micro);
    //  Deallocate Host variables
    free(a);
    free(b);
    //free(c);
    //free(d);
    free(e);
    //  Deallocate Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
    return 0;
}

__global__ void product_sumAB(int N, float *a, float *b, float *c, float *d, float *e) {
    int i = threadIdx.x;
    if (i<N) {
        c[i] = a[i]*b[i];
        d[i] = a[i+4]*b[i+4];
        e[i] = c[i]+d[i];
    }
}
