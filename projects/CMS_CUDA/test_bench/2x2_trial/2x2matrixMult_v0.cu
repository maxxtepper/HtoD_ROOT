#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nanoTime.h"

__global__ void productAB(int N, float *a, float *b, float *c);

__global__ void sumAB(int N, float *c, float *d);

__global__ void test(int N, float *c, float *d);

int main (void) {
		printf("2x2matrixMult_v0\n");
    //  Initialize parameters
    int DIM_0 = 2;
    int DIM_1 = 2;
    int N = DIM_0*DIM_1;
    size_t bytes = N*sizeof(float);
    //  Declare Host variables
    float *a, *b, *c, *d;
    //  Declare Device variables
    float *d_a, *d_b, *d_c, *d_d;
    //  Allocate Host variables
    a = (float*)malloc(bytes*2);
    b = (float*)malloc(bytes*2);
    c = (float*)malloc(bytes*2);
    d = (float*)malloc(bytes);
    //  Allocate Device variables
    hipMalloc(&d_a, bytes*2);
    hipMalloc(&d_b, bytes*2);
    hipMalloc(&d_c, bytes*2);
    hipMalloc(&d_d, bytes);
    //  Set Host variables
    a[0] = 1;
    a[1] = 1;
    a[2] = 3;
    a[3] = 3;
    a[4] = 2;
    a[5] = 2;
    a[6] = 4;
    a[7] = 4;
    b[0] = 5;
    b[1] = 6;
    b[2] = 5;
    b[3] = 6;
    b[4] = 7;
    b[5] = 8;
    b[6] = 7;
    b[7] = 8;
    //  Set Device variables
    hipMemcpy(d_a, a, bytes*2, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes*2, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, bytes*2, hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, bytes, hipMemcpyHostToDevice);
    //  Initialize Host Event Handeling
    struct timespec vartime;
    float time_elapsed_nanos;
    //  Initialize Device Event Handeling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /////////////////////////////////////////////////////////
    ////  GPU Calculation  //////////////////////////////////

		productAB<<<1,8>>>(N*2, d_a, d_b, d_c);
		sumAB<<<1,4>>>(N, d_c, d_d);

		float milli0 = 0;
		float micro0 = 0;
		float milli1 = 0;
		float micro1 = 0;
		float milli2 = 0;
		float micro2 = 0;

		hipStream_t stream0, stream1, stream2;
		hipStreamCreate(&stream0);
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipEventRecord(start,stream0);
		productAB<<<1,8, 0, stream0>>>(N*2, d_a, d_b, d_c);
		hipEventRecord(stop,stream0);
		hipStreamSynchronize(stream0);
		hipEventElapsedTime(&milli0, start, stop);
		micro0 = milli0*1000;

		hipEventRecord(start,stream1);
		sumAB<<<1,4, 0, stream1>>>(N, d_c, d_d);
		hipEventRecord(stop,stream1);
		hipStreamSynchronize(stream1);
		hipEventElapsedTime(&milli1, start, stop);
		micro1 = milli1*1000;

		hipEventRecord(start,stream2);
		test<<<1,4,0,stream2>>>(N, d_c, d_d);
		hipEventRecord(stop,stream2);
		hipStreamSynchronize(stream2);
		hipEventElapsedTime(&milli2, start, stop);
		micro2 = (milli2*1000);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipStreamDestroy(stream0);
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);

    vartime = timer_start();
		productAB<<<1,8>>>(N*2, d_a, d_b, d_c);
		sumAB<<<1,4>>>(N, d_c, d_d);
    time_elapsed_nanos = timer_end(vartime);
		float time_elapsed_micro = time_elapsed_nanos/1000;

    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Get Device variables
    hipMemcpy(d, d_d, bytes, hipMemcpyDeviceToHost);
    //  Results (GPU)
    printf("GPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Kernel Prod Elapsed Time (microseconds): %f\n", micro0);
    printf("Kernel Sum  Elapsed Time (microseconds): %f\n", micro1);
    printf("Kernel Test Elapsed Time (microseconds): %f\n", micro2);
    /////////////////////////////////////////////////////////
    ////  CPU Calculation  //////////////////////////////////
    //vartime = timer_start();
    for (int i=0; i<8; i++) c[i] = a[i]*b[i];
    for (int i=0; i<4; i++) d[i] = c[i]+c[i+4];
    //time_elapsed_nanos = timer_end(vartime);
		//float time_elapsed_micro = time_elapsed_nanos/1000;
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Results (CPU)
    //printf("CPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    //printf("Elapsed Time (microseconds): %f\n", time_elapsed_micro);
    //  Deallocate Host variables
    free(a);
    free(b);
    free(c);
    free(d);
    //  Deallocate Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    return 0;
}

__global__ void productAB(int N, float *a, float *b, float *c) {
    int i = threadIdx.x;
    if (i<N) c[i] = a[i]*b[i];
}

__global__ void sumAB(int N, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) d[i] = c[i]+c[i+4];
}

__global__ void test(int N, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) d[i] = (1/(c[i]+3)); 
}
