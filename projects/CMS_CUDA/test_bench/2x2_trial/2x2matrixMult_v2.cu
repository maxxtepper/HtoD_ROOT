#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nanoTime.h"

#define M 8

__global__ void product_sumAB(int N, float *a, float *b, float *c, float *d);

int main (void) {
    //  Initialize parameters
		printf("2x2matrixMult_v2\n");
    int DIM_0 = 2;
    int DIM_1 = 2;
    int N = DIM_0*DIM_1;
    size_t size = N*sizeof(float);
    //  Declare Host variables
    float *a, *b, *c, *d;
    //  Declare Device variables
    float *d_a, *d_b, *d_c, *d_d;
    //  Allocate Host variables
    a = (float*)malloc(size*2);
    b = (float*)malloc(size*2);
    c = (float*)malloc(size*2);
    d = (float*)malloc(size);
    //  Allocate Device variables
    hipMalloc(&d_a, size*2);
    hipMalloc(&d_b, size*2);
    hipMalloc(&d_c, size*2);
    hipMalloc(&d_d, size);
    //  Set Host variables
    a[0] = 1;
    a[1] = 1;
    a[2] = 3;
    a[3] = 3;
    a[4] = 2;
    a[5] = 2;
    a[6] = 4;
    a[7] = 4;
    b[0] = 5;
    b[1] = 6;
    b[2] = 5;
    b[3] = 6;
    b[4] = 7;
    b[5] = 8;
    b[6] = 7;
    b[7] = 8;
    //  Initialize Device Event Handeling
		float milli = 0;
		float micro = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //  Set Device variables
		hipEventRecord(start);
		hipMemcpy(d_a, a, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		micro += (milli*1000);
    //  Initialize Host Event Handeling
    struct timespec vartime;
    float time_elapsed_nanos;
    /////////////////////////////////////////////////////////
    ////  GPU Calculation  //////////////////////////////////
//		for (int i=0; i<100000; i++) {
			//  Kernel 0 (Products and Sums)
//			hipEventRecord(start);
			product_sumAB<<<1,8>>>(N, d_a, d_b, d_c, d_d);
//			hipEventRecord(stop);
//			hipEventSynchronize(stop);
			//  Device Event Results
//		}
//		micro /= 100000;
    //  Kernels Complete
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Get Device variables
    hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost);
    //  Results (GPU)
    printf("GPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Elapsed Time (microseconds): %f\n", micro);
    /////////////////////////////////////////////////////////
    ////  CPU Calculation  //////////////////////////////////
    vartime = timer_start();
    for (int i=0; i<8; i++) c[i] = a[i]*b[i];
    for (int i=0; i<4; i++) d[i] = c[i]+c[i+4];
    time_elapsed_nanos = timer_end(vartime);
		float time_elapsed_micro = time_elapsed_nanos/1000;
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Results (CPU)
    printf("CPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Elapsed Time (microseconds): %f\n", time_elapsed_micro);
    //  Deallocate Host variables
    free(a);
    free(b);
    free(c);
    free(d);
    //  Deallocate Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    return 0;
}

__global__ void product_sumAB(int N, float *a, float *b, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) {
				__shared__ float C[M];
        C[i] = a[i]*b[i];
        __syncthreads();
        if (i<4) d[i] = C[i]+C[i+4];
    }
}
