#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nanoTime.h"

__global__ void productAB(int N, float *a, float *b, float *c);

__global__ void sumAB(int N, float *c, float *d);

int main (void) {
		printf("2x2matrixMult_v0\n");
    //  Initialize parameters
    int DIM_0 = 2;
    int DIM_1 = 2;
    int N = DIM_0*DIM_1;
    size_t size = N*sizeof(float);
    //  Declare Host variables
    float *a, *b, *c, *d;
    //  Declare Device variables
    float *d_a, *d_b, *d_c, *d_d;
    //  Allocate Host variables
    a = (float*)malloc(size*2);
    b = (float*)malloc(size*2);
    c = (float*)malloc(size*2);
    d = (float*)malloc(size);
    //  Allocate Device variables
    hipMalloc(&d_a, size*2);
    hipMalloc(&d_b, size*2);
    hipMalloc(&d_c, size*2);
    hipMalloc(&d_d, size);
    //  Set Host variables
    a[0] = 1;
    a[1] = 1;
    a[2] = 3;
    a[3] = 3;
    a[4] = 2;
    a[5] = 2;
    a[6] = 4;
    a[7] = 4;
    b[0] = 5;
    b[1] = 6;
    b[2] = 5;
    b[3] = 6;
    b[4] = 7;
    b[5] = 8;
    b[6] = 7;
    b[7] = 8;
    //  Set Device variables
    hipMemcpy(d_a, a, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size*2, hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
    //  Initialize Host Event Handeling
    struct timespec vartime;
    float time_elapsed_nanos;
    //  Initialize Device Event Handeling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /////////////////////////////////////////////////////////
    ////  GPU Calculation  //////////////////////////////////
		float milli = 0;
		float micro = 0;
//		for (int i=0; i<100000; i++) {
			//  Kernel 0 (Products)
			hipEventRecord(start);
			productAB<<<1,8>>>(N*2, d_a, d_b, d_c);
			//  Kernel 1 (Sums)
			sumAB<<<1,4>>>(N, d_c, d_d);
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			//  Device Event Results
			hipEventElapsedTime(&milli, start, stop);
			micro += (milli*1000);
//		}
//		micro /= 100000;
    //  Kernels Complete
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Get Device variables
    hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost);
    //  Results (GPU)
    printf("GPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Elapsed Time (microseconds): %f\n", micro);
    /////////////////////////////////////////////////////////
    ////  CPU Calculation  //////////////////////////////////
    vartime = timer_start();
    for (int i=0; i<8; i++) c[i] = a[i]*b[i];
    for (int i=0; i<4; i++) d[i] = c[i]+c[i+4];
    time_elapsed_nanos = timer_end(vartime);
		float time_elapsed_micro = time_elapsed_nanos/1000;
    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////
    //  Results (CPU)
    printf("CPU Calculation:\n");
    //printf("%f     %f\n", d[0], d[1]);
    //printf("%f     %f\n", d[2], d[3]);
    printf("Elapsed Time (microseconds): %f\n", time_elapsed_micro);
    //  Deallocate Host variables
    free(a);
    free(b);
    free(c);
    free(d);
    //  Deallocate Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    return 0;
}

__global__ void productAB(int N, float *a, float *b, float *c) {
    int i = threadIdx.x;
    if (i<N) c[i] = a[i]*b[i];
}

__global__ void sumAB(int N, float *c, float *d) {
    int i = threadIdx.x;
    if (i<N) d[i] = c[i]+c[i+4];
}
